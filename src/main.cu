#include "hip/hip_runtime.h"
#include <boost/program_options.hpp>
#include <fstream>
#include <iostream>
#include <random>
#include <string>

#include "bfs.cuh"
#include "csr.h"
#include "cu_ef_graph.cuh"
#include "ef_layout.h"

int main(int argc, char** argv) {
    const size_t kSkipQuantum = 0;
    const size_t kForwardQuantum = 512;
    bool use_uvm;
    bool no_sort;
    uint64_t num_traversals;
    uint64_t root = std::numeric_limits<uint64_t>::max();
    std::string in_dir, mapfile;

    namespace po = boost::program_options;
    po::options_description desc(
        std::string(argv[0]) + " [OPTIONS] INPUT_DIR\nOPTIONS");
    // clang-format off
    desc.add_options()
      ("help,h", "Print help")
      ("num,n", 
       po::value(&num_traversals)->default_value(100), 
       "Number of traversals")
      ("mapfile,m",
        po::value(&mapfile),
        "Mapfile for remapping vertex ids. If provided, map[x] will "
        "be used instead of x for starting a traversal")
      ("uvm,u",
       po::bool_switch(&use_uvm)->default_value(false),
       "Use UVM for memory allocations")
      ("nosort,d",
       po::bool_switch(&no_sort)->default_value(false),
       "Disable the frontier sorting optimisation")
      ("root,r",
       po::value<uint64_t>(&root),
       "Root for the traversal. Random roots will be used if unspecified");
    // clang-format on
    po::positional_options_description pos_desc;
    po::options_description hidden;
    hidden.add_options()("in_dir,i", po::value(&in_dir));
    pos_desc.add("in_dir", -1);
    po::options_description all;
    all.add(desc).add(hidden);

    po::command_line_parser parser{argc, argv};
    parser.options(all).positional(pos_desc);
    auto parsed_options = parser.run();

    po::variables_map vm;
    po::store(parsed_options, vm);
    po::notify(vm);
    if (vm.count("help")) {
        std::cout << desc << "\n";
        return 0;
    }

    if (in_dir.empty()) {
        std::cerr << "ERROR: No input provided.\n";
        return 1;
    }

    std::cout << "Reading CSR from " << in_dir << "\n";
    CSR csr(in_dir);

    std::cout << "Total CSR size: " << csr.get_storage_size() << "\n";
    std::cout << "Optimal CSR size: " << csr.get_optimal_size() << "\n";

    EFLayout<kSkipQuantum, kForwardQuantum> ef_layout(csr);
    std::cout << "EF size: " << ef_layout.get_storage_size() << "\n";
    std::cout << "Compression ratio: "
              << csr.get_optimal_size() * 1.0 / ef_layout.get_storage_size()
              << "\n";

    std::cout << "Populating EFG Graph\n";

    CUEFGraph<kSkipQuantum, kForwardQuantum> cu_ef_graph(ef_layout, use_uvm);

    std::cout << "Graph populated\n";

    BFS<kSkipQuantum, kForwardQuantum> bfs(cu_ef_graph, !no_sort);

    std::mt19937_64 eng(0);
    std::uniform_int_distribution<uint64_t> distr;
    std::vector<uint64_t> vmap;

    if (!mapfile.empty()) {
        std::cout << "Reading mapfile " << mapfile << "\n";
        std::ifstream map_fin;
        map_fin.open(mapfile.c_str(), std::ifstream::binary);
        if (map_fin.is_open()) {
            map_fin.seekg(0, map_fin.end);
            uint64_t vertex_cnt = (map_fin.tellg() / sizeof(uint64_t));
            assert(vertex_cnt == cu_ef_graph.get_num_vertices());
            vmap.resize(vertex_cnt);
            map_fin.seekg(0, map_fin.beg);
            map_fin.read((char*)&(vmap[0]), sizeof(uint64_t) * (vertex_cnt));
            map_fin.close();
        } else {
            std::cerr << "ERROR: Could not open mapfile " << mapfile << "\n";
            return 1;
        }
    }

    std::vector<uint64_t> roots_vec(num_traversals, root);

    if (root == std::numeric_limits<uint64_t>::max()) {
        for (uint64_t i = 0; i < num_traversals; i++) {
            auto root_id = distr(eng) % cu_ef_graph.get_num_vertices();
            if (vmap.size()) root_id = vmap[root_id];
            roots_vec[i] = root_id;
        }
    }

    std::cout << "Running " << num_traversals << " traversals\n";
    // Warm Up
    bfs.traverse(roots_vec[0]);

    double total_time = 0;
    uint64_t total_visited_edges = 0;
    for (uint64_t i = 0; i < num_traversals; i++) {
        uint64_t src = roots_vec[i];
        auto distances = bfs.traverse(src);
        auto elapsed_time = bfs.get_last_elapsed_time();
        total_time += elapsed_time;
        auto visited_edges = csr.get_visited_edges(distances.data());
        total_visited_edges += visited_edges;

        if (vmap.size()) {
            std::vector<uint64_t> temp(cu_ef_graph.get_num_vertices());

            #pragma omp parallel for
            for (uint64_t j = 0; j < cu_ef_graph.get_num_vertices(); j++) {
                temp[j] = distances[vmap[j]];
            }

            distances = temp;
        }

        std::string hash = get_sha_sum(
            distances.data(), distances.size() * sizeof(distances[0]));

        std::cout << "[" << i << "]"
                  << " Source " << src << " Time " << elapsed_time
                  << " ms Visisted edges: " << visited_edges
                  << " GTEPS: " << ((double)visited_edges) * 1E-6 / elapsed_time
                  << " Hash " << hash << "\n";
    }

    std::cout << "Total time: " << total_time
              << " ms Total visited edges: " << total_visited_edges
              << " Total GTEPS: "
              << ((double)total_visited_edges) * 1E-6 / total_time << "\n";
}
